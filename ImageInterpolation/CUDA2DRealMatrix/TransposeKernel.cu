#include "hip/hip_runtime.h"
#include "TransposeKernel.h"

__global__ void TransposeAndCastComplexSingleToDouble(hipDoubleComplex * odata, hipComplex *idata, unsigned long width, unsigned long height) {

	unsigned long int xIndex     = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned long int yIndex     = blockIdx.y * blockDim.y + threadIdx.y;

	if (xIndex < width && yIndex < height) {

		hipComplex inTemp = idata[yIndex * width + xIndex];
		hipDoubleComplex outVal;
		outVal.x                        = (double) inTemp.x;
		outVal.y                        = (double) inTemp.y;
		odata[xIndex * height + yIndex] = outVal;
	}
}

__global__ void TransposeAndCastComplexDoubleToSingle(hipComplex * odata, hipDoubleComplex *idata, unsigned long width, unsigned long height) {

	unsigned long int xIndex     = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned long int yIndex     = blockIdx.y * blockDim.y + threadIdx.y;

	if (xIndex < width && yIndex < height) {

		hipDoubleComplex inTemp = idata[yIndex * width + xIndex];
		hipComplex outVal;
		outVal.x                        = (float) inTemp.x;
		outVal.y                        = (float) inTemp.y;
		odata[xIndex * height + yIndex] = outVal;
	}
}